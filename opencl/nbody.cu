#include "hip/hip_runtime.h"
#include "cosmos.cu"

//utility I/O functions for reading in Cosmoses
//gets the next good line, checking for EOF and ignoring whitespace and commented lines
int getnextgoodline( char * s, int n, FILE * f){
  fgets(s, n, f);
  if(NULL == s){
    return 1;
    //semicolon escapes commented lines
  }else if(';' == s[0] || '\n' == s[0]){
    return getnextgoodline( s, n, f);    
  }
  return 0;
}

//splits a string into chunks based on the delimiter, resultc is max results
//returns number of results retrieved
int chunk_string(char * s, char * delim, int resultc, char ** res){
  if(0 == resultc || '\n' == s[0] || '\0' == s[0]){
    return 0;
  }
  int ch_cnt = strcspn( s, delim );
  res[0] = (char *) malloc( (ch_cnt + 1) * sizeof(char));
  if(NULL == res[0]){
    printf("error in malloc\n");
    exit(0);
  }
  strncpy(res[0], s, (ch_cnt));
  res[0][ch_cnt] = '\0';
  if('\0' == s[ch_cnt] || '\n' == s[ch_cnt] ){
    return 1;
  } 
  return 1 + chunk_string( &(s[ch_cnt + 1]), delim, --resultc, &(res[1]));
}

//converts a string to a floating point number
float string_to_float( char * s ){
  char * nums[2];
  int resct = chunk_string( s, ".", 2, nums);
  if(0 == resct || 2 < resct){
    printf("string: \"%s\" is not a float!\n", s);
    return 0.0;
  }
  if(1 == resct){
    return (float)atol(nums[0]);
  }
  return (float)atol(nums[0]) + (atol(nums[1]) / pow(10, (int)strlen(nums[1])) ) ;
}

//frees chunk-based arrays
void free_chunks(char ** c, int num){
  
  for(int i = 0; i < num; i++){
    free(c[i]);
  }

  return;
}


struct body * get_body_host(struct cosmos * c, int k){
  return (&(c->body_list[k]));
}

//cosmos_free frees a cosmos
void cosmos_free( struct cosmos * c ){
    //free each body first
  free(c->body_list);
  free(c);
  return;
}


void read_body( struct body * b, char * s){
  char * chunk_res[3];
  int num_chunks = chunk_string(s , " " , 3, chunk_res);
  if(3 != num_chunks){
    printf("string \"%s\" is not a valid body!\n", s);
  }

  /*the first number represents mass, the second x posn, the third y posn
  all objects start with zero velocities. All lines starting with semicolons
  are ignored. 
  */
  
  b->mass = string_to_float( chunk_res[0] );
  b->x_posn = string_to_float( chunk_res[1] );
  b->y_posn = string_to_float( chunk_res[2] );
  b->x_velocity = 0.0;
  b->y_velocity = 0.0;
  b->x_force = 0.0;
  b->y_force = 0.0;
  free_chunks(chunk_res, 3);
  return;
}

//cosmos_init initializes a cosmos with parameters as given
struct cosmos * cosmos_init( float xsize, float ysize, float ts, int numbodies){

  struct cosmos * p;
  p = (struct cosmos *) malloc(sizeof(struct cosmos));
  if(p == NULL){
    printf("error in malloc");
    exit(1);
  }
  p->num_bodies = numbodies;
  p->time_step = ts;
  p->x_world_size = xsize;
  p->y_world_size = ysize;
  p->body_list = (struct body *)malloc(numbodies * sizeof(struct body));
  if(p->body_list == NULL){
    printf("error in malloc");
    exit(1);
  }
  
  return p;

}

//get_bodies fills the cosmos with bodies.
//the bodies it fills it with are "dummy" bodies" all at the same place,
//the same mass, etc. 
struct cosmos * get_cosmos( int n){
  struct cosmos * c;
  c = cosmos_init(0.5, n, 500, 500); 


  for(int i = 0; i < n; i++){

    struct body * b;
    b = get_body_host(c,i);
    //these are filler values
    b->x_posn = 2.0;
    b->y_posn = 2.0;
    b->mass = 1.0;
    b->x_velocity = 0.0;
    b->y_velocity = 0.0;
  }
   
  return c;

}

//takes a filename, gets a cosmos, viola!
struct cosmos * read_cosmos( const char * filename ) {

  FILE * fp;
  int num_res;
  char * firstlineargs[4];
  fp = fopen( filename, "r" );

  if(NULL == fp){
    printf("unable to open file %s \n", filename);
    exit(1);
  }

  char firstline[80];
  getnextgoodline(firstline, 79, fp);
  if( NULL == firstline || ':' != firstline[0]){
    printf("test file in wrong format\n");
    return NULL;
  }
  num_res = chunk_string( &(firstline[1]), " ", 4 , firstlineargs );
  if(4 != num_res){
    printf("wrong number of arguments on input file!\n");
    return NULL;
  }

  struct cosmos * c = 
    cosmos_init(string_to_float( firstlineargs[0] ), 
		string_to_float( firstlineargs[1] ), 
		string_to_float( firstlineargs[2] ), 
		atoi( firstlineargs[3] ) );

  free_chunks(firstlineargs, 4);

  char line[100];
  for(int i = 0; i < c->num_bodies && (0 == getnextgoodline(line, 99, fp)); i++){
    read_body(  &(c->body_list[i]) , line);
  }
  
  return c;
}


//prints a body
void print_body( struct body * b){
  
  printf("body x-position: %f, body y-position: %f\n", b->x_posn, b->y_posn);
  printf("body mass: %f\n", b->mass);

  return;
}

//prints the given cosmos;
void print_cosmos( struct cosmos * c ){
  
  printf("cosmos x-size: %f, cosmos y-size: %f\n", c->x_world_size, c->y_world_size);
  printf("time step: %f\n", c->time_step);
  printf("number of bodies: %i\n", c->num_bodies);
  for(int i = 0; i < c->num_bodies; i++){
    print_body( get_body_host(c,i) );
  }
  return;

}

//please note that this code has NEVER been compiled or tested
int main(){
// create OpenCL device & context
cl_context hContext;
hContext = clCreateContextFromType(0, CL_DEVICE_TYPE_GPU,
                                   0, 0, 0);

// query all devices available to the context
size_t nContextDescriptorSize;
clGetContextInfo(hContext, CL_CONTEXT_DEVICES,
                 0, 0, &nContextDescriptorSize);
cl_device_id * aDevices = (cl_device_id *)malloc(nContextDescriptorSize);
clGetContextInfo(hContext, CL_CONTEXT_DEVICES,
                 nContextDescriptorSize, aDevices, 0);


const char **kernelcode = {"copy nbody.cl into this string"};
  // create OpenCL device & context
  cl_context hContext;
  hContext = clCreateContextFromType(0, CL_DEVICE_TYPE_GPU,
                                   0, 0, 0);

  // query all devices available to the context
  size_t nContextDescriptorSize;
  clGetContextInfo(hContext, CL_CONTEXT_DEVICES,
                 0, 0, &nContextDescriptorSize);
  cl_device_id * aDevices = (cl_device_id *)malloc(nContextDescriptorSize);
  clGetContextInfo(hContext, CL_CONTEXT_DEVICES,
                 nContextDescriptorSize, aDevices, 0);

  cl_program hProgram;
  hProgram = clCreateProgramWithSource(hContext, 1,
                                     sProg, 0, 0);

  clBuildProgram(hProgram, 
	0, 
	0, 
	0, 
	0, 
	0);


  // create kernel
  cl_kernel krn;
  krn = clCreateKernel(hProgram, 
	"void nbodycompute", 
	0);
  cl_command_queue hCmdQueue;
  CmdQueue = clCreateCommandQueue(hContext, aDevices[0], 0, 0);

  struct cosmos * c = read_cosmos("test.dat");
  struct cosmos * b_res = malloc(sizeof(struct body) * c->num_bodies);

  cl_mem c_d  = clCreateBuffer(hContext,
			       CL_MEM_READ_WRITE | CL_MEM_COPY_HOST_PTR,
                               sizeof(struct cosmos),
                               c,
			       NULL); 

  cl_mem b_d = clCreateBuffer(hContext,
                              CL_MEM_READ_WRITE | CL_MEM_COPY_HOST_PTR,
                              sizeof(struct body) * c->num_bodies,
                              b,
			      NULL);
  
  clSetKernelArg(krn, 0, sizeof(cl_mem), (void *) &c_d)
  clSetKernalArg(krn, 1, sizeof(cl_mem), (void *) &b_d)

  clEnqueueNDRangeKernel(CmdQueue, krn,
			 1, //work dimension
			 0, //global work offset
                         c->num_bodies, //global work size = number of work-items (i.e. threads) in work_dim dimensions that execute the kernel function
                         0, //local work size
                         0, //number of waiting events
                         0, //list of waiting events
                         0); //pointer to some event

//since the world isn't altered, we just need to read the bodies back
  clEnqueueReadBuffer(CmdQueue, b_d, CL_TRUE, 0,
			sizeof(struct body) * c->num_bodies , //size of memory to be copied
			b_res, 0, 0, 0); //more waiting events stuff
 
  c->body_list = b_res;
  print_cosmos(c);
  
  free_cosmos(c);
  clReleaseMemObj(c_d);
  clReleaseMemObj(b_d);

return 0;
}

