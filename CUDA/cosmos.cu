#include "hip/hip_runtime.h"
#include "cosmos.h"


//utility I/O functions for reading in Cosmoses
//gets the next good line, checking for EOF and ignoring whitespace and commented lines
int getnextgoodline( char * s, int n, FILE * f){
  fgets(s, n, f);
  if(NULL == s){
    return 1;
    //semicolon escapes commented lines
  }else if(';' == s[0] || '\n' == s[0]){
    return getnextgoodline( s, n, f);    
  }
  return 0;
}

//splits a string into chunks based on the delimiter, resultc is max results
//returns number of results retrieved
int chunk_string(char * s, char * delim, int resultc, char ** res){
  if(0 == resultc || '\n' == s[0] || '\0' == s[0]){
    return 0;
  }
  int ch_cnt = strcspn( s, delim );
  res[0] = (char *) malloc( (ch_cnt + 1) * sizeof(char));
  if(NULL == res[0]){
    printf("error in malloc\n");
    exit(0);
  }
  strncpy(res[0], s, (ch_cnt));
  if('\0' == s[ch_cnt] || '\n' == s[ch_cnt] ){
    return 1;
  } 
  return 1 + chunk_string( &(s[ch_cnt + 1]), delim, --resultc, &(res[1]));
}

//converts a string to a floating point number
float string_to_float( char * s ){
  char * nums[2];
  int resct = chunk_string( s, ".", 2, nums);
  if(0 == resct || 2 < resct){
    printf("string: \"%s\" is not a float!\n", s);
    return 0.0;
  }
  if(1 == resct){
    return (float)atol(nums[0]);
  }
  return (float)atol(nums[0]) + (atol(nums[1]) / pow(10, (int)strlen(nums[1])) ) ;
}

//frees chunk-based arrays
void free_chunks(char ** c, int num){
  
  for(int i = 0; i < num; i++){
    free(c[i]);
  }

  return;
}


//returns body k in cosmos c
__device__ struct body * get_body(struct cosmos * c, int k){
  return (&(c->body_list[k]));
}

struct body * get_body_host(struct cosmos * c, int k){
  return (&(c->body_list[k]));
}

//cosmos_free frees a cosmos
void cosmos_free( struct cosmos * c ){
    //free each body first
  free(c->body_list);
  free(c);
  return;
}


void read_body( struct body * b, char * s){
  char * chunk_res[3];
  int num_chunks = chunk_string(s , " " , 3, chunk_res);
  if(3 != num_chunks){
    printf("string \"%s\" is not a valid body!\n", s);
  }

  /*the first number represents mass, the second x posn, the third y posn
  all objects start with zero velocities. All lines starting with semicolons
  are ignored. 
  */
  
  b->mass = string_to_float( chunk_res[0] );
  b->x_posn = string_to_float( chunk_res[1] );
  b->y_posn = string_to_float( chunk_res[2] );
  
  free_chunks(chunk_res, 3);
  return;
}

//get_bodies fills the cosmos with bodies.
//the bodies it fills it with are "dummy" bodies" all at the same place,
//the same mass, etc. 
struct cosmos * get_cosmos( int n){
  struct cosmos * c;
  c = cosmos_init(0.5, n, 500, 500); 


  for(int i = 0; i < n; i++){

    struct body * b;
    b = get_body_host(c,i);
    //these are filler values
    b->x_posn = 2.0;
    b->y_posn = 2.0;
    b->mass = 1.0;
    b->x_velocity = 0.0;
    b->y_velocity = 0.0;
  }
   
  return c;

}

//cosmos_init initializes a cosmos with parameters as given
struct cosmos * cosmos_init( float xsize, float ysize, float ts, int numbodies){

  struct cosmos * p;
  p = (struct cosmos *) malloc(sizeof(struct cosmos));
  if(p == NULL){
    printf("error in malloc");
    exit(1);
  }
  p->num_bodies = numbodies;
  p->time_step = ts;
  p->x_world_size = xsize;
  p->y_world_size = ysize;
  p->body_list = (struct body *)malloc(numbodies * sizeof(struct body));
  if(p->body_list == NULL){
    printf("error in malloc");
    exit(1);
  }
  
  return p;

}

//takes a filename, gets a cosmos, viola!
struct cosmos * read_cosmos( const char * filename ) {

  FILE * fp;
  int num_res;
  char * firstlineargs[4];
  fp = fopen( filename, "r" );

  if(NULL == fp){
    printf("unable to open file %s \n", filename);
    exit(1);
  }

  char firstline[80];
  getnextgoodline(firstline, 79, fp);
  if( NULL == firstline || ':' != firstline[0]){
    printf("test file in wrong format\n");
    return NULL;
  }
  num_res = chunk_string( &(firstline[1]), " ", 4 , firstlineargs );
  if(4 != num_res){
    printf("wrong number of arguments on input file!\n");
    return NULL;
  }

  struct cosmos * c = 
    cosmos_init(string_to_float( firstlineargs[0] ), 
		string_to_float( firstlineargs[1] ), 
		string_to_float( firstlineargs[2] ), 
		atoi( firstlineargs[3] ) );

  free_chunks(firstlineargs, 4);

  char line[100];
  for(int i = 0; i < c->num_bodies && (0 == getnextgoodline(line, 99, fp)); i++){
    read_body(  &(c->body_list[i]) , line);
  }
  
  return c;
}



//prints a body
void print_body( struct body * b){
  
  printf("body x-position: %f, body y-position: %f\n", b->x_posn, b->y_posn);
  printf("body mass: %f\n", b->mass);

  return;
}

//prints the given cosmos;
void print_cosmos( struct cosmos * c ){
  
  printf("cosmos x-size: %f, cosmos y-size: %f\n", c->x_world_size, c->y_world_size);
  printf("time step: %f\n", c->time_step);
  printf("number of bodies: %i\n", c->num_bodies);
  for(int i = 0; i < c->num_bodies; i++){
    print_body( get_body_host(c,i) );
  }
  return;

}

__device__ float sign( float f ){
  if (f == 0.0){
    return 0.0;
  }
  if (f < 0.0){
    return -1.0;
  }
  return 1.0;
}

__device__ void point_update_compute(struct body * b1, struct body * b2, float gravity){

  float xdiff = b1->x_posn - b2->x_posn;
  float ydiff = b1->y_posn - b2->y_posn;
  float dist_sq = (xdiff*xdiff) + (ydiff*ydiff);
  float force = gravity * b1->mass * b2->mass / dist_sq;
  float theta = atan(ydiff/xdiff);
  b1->x_force += force * sign(xdiff) * cos(theta);
  b1->y_force += force * sign(ydiff) * sin(theta);
  //may want these lines to be "+=" we'll find out
  b2->x_force -= force * sign(xdiff) * cos(theta);
  b2->y_force -= force * sign(ydiff) * sin(theta);

  return;
}


__device__ void calc_movement(struct body * b,const float time_step){

  //velocity = acceleration * time
  //v = v_0 + a * t
  b->x_velocity += (b->x_force / b->mass) * time_step;
  b->y_velocity += (b->y_force / b->mass) * time_step;

  //may need to have this divided by time_step, not sure
  //x = x_0 + v * t
  //the '-' here is a hack because we seem to be getting the wrong numbers
  b->x_posn = b->x_posn - (b->x_velocity * time_step);
  //or maybe the '+' here is the hack. We'll find out
  b->y_posn = b->y_posn + (b->y_velocity * time_step);

  return;
}

