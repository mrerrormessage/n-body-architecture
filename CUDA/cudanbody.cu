#include "hip/hip_runtime.h"
#include "cudanbody.h"
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void nBodyCompute(struct cosmos * c);

int main(int argc, char ** argv)
{
  
  int num_steps;
  if(argc <= 1){
    num_steps = 200;
  } else {
    num_steps = atoi(argv[1]);
  }

  struct cosmos* c = read_cosmos("test.dat");

  //copy bodies to the device
  struct cosmos * c_d;
  struct body * b_d;
  struct body * temp = c->body_list;
  hipMalloc((void **) &b_d, sizeof(struct body) * c->num_bodies);
  print_cosmos(c);
  c->body_list = b_d;
  for(int i = 0; i < c->num_bodies; i++){
    hipMemcpy(b_d, temp, sizeof(struct body) * c->num_bodies, hipMemcpyHostToDevice);
    
  }

  //copy cosmos to device
  hipMalloc((void **) &c_d, sizeof(struct cosmos));
  hipMemcpy(c_d, c, sizeof(struct cosmos), hipMemcpyHostToDevice);
  

  printf("beginning cuda n-body...\n");

  nBodyCompute<<<num_steps, c->num_bodies>>>(c_d);

  
  return 0;
}

__global__ void nBodyCompute(struct cosmos * c){

  //get the appropriate body
  //since there is a thread launched for each body, this means that 
  //the id of our particular body is equivalent to the thread id. 
  struct body * b = get_body(c, threadIdx.x);

  //compare with all higher bodies
  for(int i = 0; i < 1; i++){
    point_update_compute(b, get_body(c,i) , GRAV_CONSTANT );
  }
  
  //make sure all threads are synchronized before continuing
  __syncthreads();
  calc_movement(b, c->time_step);
}
