#include "hip/hip_runtime.h"
#include "cudanbody.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void nBodyCompute(struct cosmos * c, struct body * b);

int main(int argc, char ** argv)
{
  
  hipSetDevice(0);
  hipSetDeviceFlags(hipDeviceMapHost);
  int num_steps;
  if(argc <= 1){
    num_steps = 200;
  } else {
    num_steps = atoi(argv[1]);
  }

/*  struct cosmos * c, * c_d;
  struct body * b, * b_d;
  hipHostAlloc((void **)&c, sizeof(struct cosmos), hipHostMallocMapped);
  c = read_cosmos("test.dat");
  hipHostAlloc((void **)&b, sizeof(struct body) * c->num_bodies, hipHostMallocMapped);
  for(int i = 0; i < c->num_bodies; i++){
    memcpy(b, c->body_list, sizeof(struct body) * c->num_bodies);
  }

  print_cosmos(c);

  hipHostGetDevicePointer((void **)&b_d, (void *)b, 0);
  hipHostGetDevicePointer((void **)&c_d, (void *)c, 0);

*/
  //copy bodies to the device

  struct cosmos * ret_c = (struct cosmos *) malloc(sizeof(struct cosmos));
  struct cosmos * c = read_cosmos("test.dat");
  struct body * ret_b = (struct body *) malloc(sizeof(struct body) * c->num_bodies);
  struct cosmos * c_d; 
  struct body * b_d;
  

  hipMalloc((void **) &b_d, sizeof(struct body) * c->num_bodies);
  hipMalloc((void **) &c_d, sizeof(struct cosmos));
  print_cosmos(c);
/*
  const float test_f[] = {1.0};
  float * test_f_p = (float *) malloc(sizeof(float));
  float * test_f_d; 
  hipMalloc((void **) &test_f_d, sizeof(float));
  hipMemcpy((void *)test_f_d, (void *)test_f, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy((void *)test_f_p, (void *)test_f_d, sizeof(float), hipMemcpyDeviceToHost);
  printf("test returned: %f\n", test_f_p[0]);
*/

//  for(int i = 0; i < c->num_bodies; i++){
  hipMemcpy(b_d, c->body_list, sizeof(struct body) * c->num_bodies, hipMemcpyHostToDevice);    
//  }

  c->body_list = b_d;
  //copy cosmos to device
  hipMemcpy(c_d, c, sizeof(struct cosmos), hipMemcpyHostToDevice);  
  
  printf("beginning cuda n-body...\n");

  nBodyCompute<<<num_steps, c->num_bodies>>>(c_d, b_d);
//  hipDeviceSynchronize();
  printf("after computation...\n");

  //once we return, we need to copy the memory back and examine it. 
  hipMemcpy(ret_c, c_d, sizeof(struct cosmos), hipMemcpyDeviceToHost);
  printf("copied cosmos, %i bodies\n", ret_c->num_bodies);

  hipMemcpy(ret_b, b_d, sizeof(struct body) * ret_c->num_bodies, hipMemcpyDeviceToHost);
  printf("copied bodies\n");
  ret_c->body_list = ret_b;
  print_cosmos(ret_c);


/*
  printf("after computation\n");

  //once we return, we need to copy the memory back and examine it. 
  hipMemcpy(ret_c, c_d, sizeof(struct cosmos), hipMemcpyDeviceToHost);
  printf("copied cosmos, %i bodies\n", ret_c->num_bodies);

  hipMemcpy(ret_b, b_d, sizeof(struct body) * ret_c->num_bodies, hipMemcpyDeviceToHost);
  printf("copied bodies\n");

  ret_c->body_list = ret_b; */
  hipFree(b_d);
  hipFree(c_d);
  hipDeviceReset();

  return 0;
}

__global__ void nBodyCompute(struct cosmos * c, struct body * blist){
 
  int i_am = threadIdx.x;
  struct body * b = &(blist[i_am]);

  //compare with all higher bodies
  for(int i = i_am + 1; i < c->num_bodies; i++){
    point_update_compute(b, &(blist[i]) , GRAV_CONST );
  }
  
  //make sure all threads are synchronized before continuing
  __syncthreads();
  calc_movement(b, c->time_step); 
}


